#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include "highgui.h"
#include "imgproc.h"
#include "ovr.h"
#include "gcube.h"

#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include <fcntl.h>
#include <string.h>
#include <string>
#include <signal.h>
#include "shmdata.h"
#include "color.h"
#include "gcube.h"
#include "gpu_util.h"

using namespace std;

void preprocessSHM(int *img, gcube &bgr);
void subplace(gcube &subimage, gcube &orig, int left, int top);

static int stopsig;
void stopme(int signo) {
  stopsig = 1;
}

int main(int argc, const char *argv[]) {
  signal(SIGINT, stopme);

  key_t key;
  int shmflg;
  int shmid;
  int size;

  key = 9000;
  size = sizeof(shmdata);
  shmflg = 0666;

  // get an shm
  if ((shmid = shmget(key, size, shmflg)) == -1) {
    print_error("[SINK] shmget failed");
    return 1;
  } else {
    print_debug("[SINK] Got an shm!");
  }

  // attach the shm to this process
  shmdata *data = NULL;
  data = (shmdata *)shmat(shmid, data, shmflg);
  if (data == (shmdata *)-1) {
    print_error("[SINK] failed to attach");
    return 1;
  } else {
    print_debug("[SINK] found character stream");
  }
  // END SHM

  // Copy into gpu
  int *img;
  checkCudaErrors(hipMalloc((void **)&img, SHMDATASIZE));

  /** this operation should do:
   *  1) restrict range
   *  2) convert the color
   *  3) possibly flip depending on if dk1 or dk2
   */

#if DK2
  gcube bgr(data->height, data->width - 1920, 3);
#else
  gcube bgr(data->height, data->width - 1280, 3); // removes the excess width from DK1
#endif

  /** get the subimages
   */
  size_t new_width = 500;
  size_t new_height = new_width * bgr.n_rows / bgr.n_cols;
  size_t crop = 20;//new_width / 20;
  size_t subimage_width = new_width - crop;
  size_t subimage_height = new_height;

#if DK2
  size_t xoffset = 960 - subimage_width;
  size_t yoffset = 540 - (subimage_height / 2) - 25;

  gcube limg(1080, 960, 3, gfill::zeros);
  gcube rimg(1080, 960, 3, gfill::zeros);
  gcube combined(1080, 1920, 3, gfill::zeros);

#else

  size_t xoffset = 640 - subimage_width;
  size_t yoffset = 400 - (subimage_height / 2) - 25; // hacked 25 extra offset up

  gcube limg(800, 640, 3, gfill::zeros);
  gcube rimg(800, 640, 3, gfill::zeros);
  gcube combined(800, 1280, 3, gfill::zeros);
#endif

  double offset = 0.15;

  while (!stopsig) {
    // copy to gpu over from the shm
    checkCudaErrors(hipMemcpy(img, data->data, SHMDATASIZE, hipMemcpyHostToDevice));

    // do triple operation
    preprocessSHM(img, bgr);

    // resize the gcube
    gcube resized = gpu_imresize2(bgr, new_height, new_width);

    // after resize, then place the resized image onto two subimages
    subplace(limg, resized, xoffset, yoffset);
    subplace(rimg, resized, -crop, yoffset);

    combined = ovr_image(limg, rimg, offset);
    cv::Mat out = combined.cv_img();
    cv::imshow("hud", out);
    if (cv::waitKey(30) >= 0) {
      continue;
    }

    // reset the cuda buffer
    checkCudaErrors(hipMemset(limg.d_pixels, 0, limg.n_elem * sizeof(float)));
    checkCudaErrors(hipMemset(rimg.d_pixels, 0, rimg.n_elem * sizeof(float)));
    checkCudaErrors(hipMemset(combined.d_pixels, 0, combined.n_elem * sizeof(float)));
  }

  return 0;
}

__global__ void GPU_preprocess(float *bgr, uint8_t *img, int n_rows, int src_cols, int dst_cols) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= n_rows || j >= dst_cols) {
    return;
  }
  // row-wise
  bgr[IJK2C(i, j, 0, n_rows, dst_cols)] = (float)img[IJK2C(3-0-1, j, i, 4, src_cols)] / 255.0;
  bgr[IJK2C(i, j, 1, n_rows, dst_cols)] = (float)img[IJK2C(3-1-1, j, i, 4, src_cols)] / 255.0;
  bgr[IJK2C(i, j, 2, n_rows, dst_cols)] = (float)img[IJK2C(3-2-1, j, i, 4, src_cols)] / 255.0;
}

void preprocessSHM(int *img, gcube &bgr) {
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((bgr.n_rows-1)/16+1, (bgr.n_cols-1)/16+1, 1); // range restricted, adds back excess for calculation
#if DK2
  size_t OVRCOLS = 1920;
#else
  size_t OVRCOLS = 1280;
#endif
  GPU_preprocess<<<gridSize, blockSize>>>(bgr.d_pixels, (uint8_t *)img, bgr.n_rows, bgr.n_cols + OVRCOLS, bgr.n_cols);
  checkCudaErrors(hipGetLastError());
}

__global__ void GPU_subplace(float *dst, float *src, int left, int top, int dw, int dh, int sw, int sh) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (top + i < 0 || top + i >= dh || left + j < 0 || left + j >= dw || i >= sh || j >= sw) {
    return;
  }
  dst[IJK2C(top + i, left + j, 0, dh, dw)] = src[IJK2C(i, j, 0, sh, sw)];
  dst[IJK2C(top + i, left + j, 1, dh, dw)] = src[IJK2C(i, j, 1, sh, sw)];
  dst[IJK2C(top + i, left + j, 2, dh, dw)] = src[IJK2C(i, j, 2, sh, sw)];
}

void subplace(gcube &subimage, gcube &orig, int left, int top) {
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((orig.n_rows-1)/16+1, (orig.n_cols-1)/16+1, 1);
  GPU_subplace<<<gridSize, blockSize>>>(subimage.d_pixels, orig.d_pixels, left, top, subimage.n_cols, subimage.n_rows, orig.n_cols, orig.n_rows);
  checkCudaErrors(hipGetLastError());
}
