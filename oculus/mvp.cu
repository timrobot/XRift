#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include "highgui.h"
#include "ovr.h"
#include "gcube.h"

#include <sys/types.h>
#include <sys/ipc.h>
#include <sys/shm.h>
#include <cstdio>
#include <cstdlib>
#include <unistd.h>
#include "shmdata.h"
#include "color.h"
#include "gcube.h"
//#include "gpu_util.h"

using namespace std;

void preprocessSHM(int *img, gcube &bgr);
void subplace(gcube &subimage, gcube &orig, int left, int top, int width, int height);

static int stopsig;
void stopme(int signo) {
  stopsig = 1;
}

int main(int argc, const char *argv[]) {
  signal(SIGINT, stopme);

  key_t key;
  int shmflg;
  int shmid;
  int size;

  key = 9000;
  size = sizeof(shmdata);
  shmflg = 0666;

  // get an shm
  if ((shmid = shmget(key, size, shmflg)) == -1) {
    print_error("[SINK] shmget failed");
    return 1;
  } else {
    print_debug("[SINK] Got an shm!");
  }

  // attach the shm to this process
  shmdata *data = (shmdata *)shmat(shmid, data, shmflg);
  if (data == (shmdata *)-1) {
    print_error("[SINK] failed to attach");
    return 1;
  } else {
    print_debug("[SINK] found character stream");
  }
  // END SHM

  // Copy into gpu
  int *img;
  checkCudaErrors(hipMalloc(&img, SHMDATASIZE));

  /** this operation should do:
   *  1) restrict range
   *  2) convert the color
   *  3) possibly flip depending on if dk1 or dk2
   */
  gcube bgr(data->height, data->width - 1280, 3); // removes the excess width from DK1

  /** get the subimages
   */
  size_t new_width = 500;
  size_t new_height = new_width * bgr.n_rows / bgr.n_cols;
  size_t crop = new_width / 20;
  size_t subimage_width = new_width - crop;
  size_t subimage_height = new_height;

  size_t xoffset = 640 - subimage_width;
  size_t yoffset = 400 - (subimage_height / 2);

  gcube limg(800, 640, 3, fill::zeros);
  gcube rimg(800, 640, 3, fill::zeros);
  gcube combined(800, 1280, 3, fill::zeros);

  while (!stopsig) {
    // copy to gpu over from the shm
    checkCudaErrors(hipMemcpy(img.d_pixels, data->data, SHMDATASIZE, hipMemcpyHostToDevice));

    // do triple operation
    preprocessSHM(img, bgr);

    // resize the gcube
    gcube resized = gpu_imresize2(bgr, new_width, new_height);

    // after resize, then place the resized image onto two subimages
    subplace(limg, resized, xoffset, yoffset, subimage_width, subimage_height);
    subplace(rimg, resized, 0, yoffset, subimage_width, subimage_height);

    combined = ovr_image(limg, rimg, offset);
    cv::Mat out = combined.cv_img();
    cv::imshow("hud", out);
    if (cv::waitKey(30) >- 0) {
      continue;
    }
  }

  return 0;
}

__global__ void GPU_preprocess(float *bgr, uint8_t *img, int n_rows, int src_cols, int dst_cols) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= n_rows || j >= dst_cols) {
    return;
  }
  // row-wise
  int src_idx = 4 * (i * src_cols + j);
  int dst_idx = 3 * (i * dst_cols + j);
  bgr[dst_idx + 0] = float(img[src_idx + 0]) / 255.0;
  bgr[dst_idx + 1] = float(img[src_idx + 1]) / 255.0;
  bgr[dst_idx + 2] = float(img[src_idx + 2]) / 255.0;
}

void preprocessSHM(int *img, gcube &bgr) {
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((bgr.n_rows-1)/16+1, (bgr.n_cols-1)/16+1, 1); // range restricted
  GPU_preprocess<<<gridSize, blockSize>>>(bgr.d_pixels, (uint8_t *)img, bgr.n_rows, bgr.n_cols, bgr.n_cols - 1280);
  checkCudaErrors(cudaGetLastErrror());
}

__global__ void GPU_subplace(float *dst, float *src, int left, int top, int sw, int sh, int width, int height) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= sh || j >= sw) {
    return;
  }
  // row-wise
  int src_idx = 3 * (i * width + j);
  int dst_idx = 3 * ((top + i) * sw + (left + j));
  dst[dst_idx + 0] = src[src_idx + 0];
  dst[dst_idx + 1] = src[src_idx + 1];
  dst[dst_idx + 2] = src[src_idx + 2];
}

void subplace(gcube &subimage, gcube &orig, int left, int top, int width, int height) {
  dim3 blockSize(16, 16, 1);
  dim3 gridSize((height-1)/16+1, (width-1)/16+1, 1);
  GPU_subplace<<<gridSize, blockSize>>>(subimage.d_pixels, orig.d_pixels, left, top, subimage.n_cols, subimage.n_rows, width, height);
  checkCudaErrors(hipGetLastError());
}
