#include "hip/hip_runtime.h"
#include "highgui.h"
#include "imgproc.h"
#include "gpu_util.h"

//const float u_distortion[4] = { 1.0, -0.22, -0.24, 0 };
// preprogrammed constants
#define UD0 1.0
#define UD1 0.22
#define UD2 0.24

// This GPU function is used inside the CPU function ovr_image(const gcube &, const gcube &, double)
__global__ void barrel_distort_ovr(float *G, float *F,
    int n_rows, int n_cols, int n_slices,
    float r_x, float r_y, float r_max, float offset_x, int right_image) {

  /** This kernel is like a GPU-parallelized for-loop:

    for (int i = 0; i < rows; i++) {
      for (int j = 0; j < cols; j++) {
        // GPU code here
      }
    }

  */

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y * blockDim.y + threadIdx.y;

  // Everything below is like the block of code inside a parallelized GPU nested for-loop

  if (i >= n_rows || j >= n_cols) {
    return;
  }
  float x = (float)(j-r_x) / r_max + offset_x;
  float y = (float)(i-r_y) / r_max;
  // distortionScale
  float rr = x*x+y*y;
  float distortion = UD0+UD1*rr+UD2*rr*rr;
  int _i = (int)roundf(distortion*y*r_max+r_y);
  int _j = (int)roundf((distortion*x-offset_x)*r_max+r_x);
  for (int k = 0; k < n_slices; k++) {
    // since they are stored right next to each other, the slices are the separating factor (k*2+right_image)
    if (_i >= 0 && _i < n_rows && _j >= 0 && _j < n_cols) {
      G[IJK2C(i, j, k*2+right_image, n_rows, n_cols)] = F[IJK2C(_i, _j, k, n_rows, n_cols)];
    }
  }
}

/** This function takes in 2 gcubes (a GPU-enabled image), and outputs another gcube which is the oculus-transformed image
 *  @param left the left image
 *  @param right the right image
 *  @param offset_x the calibration constant for the offset of the rift headset, manually inputted (try 0.15)
 *  @return a new image (oculus-displayable)
 */
gcube ovr_image(const gcube &left, const gcube &right, double offset_x) {
  // assume that the left and right images have equal dimensions
  assert(left.n_rows == right.n_rows && left.n_cols == right.n_cols);

  // find the middle of the left image's width + (1 + |offset_x|)
  double mrx = left.n_cols / 2.0 * (1 + abs(offset_x));

  // find the middle of the right image's width
  double mry = right.n_cols / 2.0;

  // find the distortion radius
  float r_max = (float)sqrt(mrx * mrx + mry * mry);

  // find the middle point for either image
  float r_x = left.n_cols / 2.0f;
  float r_y = left.n_rows / 2.0f;

  // define the dimensions of the blockSize (GPU-only)
  dim3 blockSize(16, 16, 1);

  // define the dimensions of the gridSize (GPU-only)
  dim3 gridSize((left.n_cols-1)/16+1, (left.n_rows-1)/16+1, 1);

  // create a <dst> GCube (this is a custom class that I defined somewhere
  // with height = left.height, width = left.width + right.width, channels = left.n_channels, filled with 0s
  gcube C(left.n_rows, left.n_cols + right.n_cols, left.n_slices, gfill::zeros);

  // use the GPU to barrel distort the source images (left and right) into the destination (gcube C)
  barrel_distort_ovr<<<gridSize, blockSize>>>(C.d_pixels, left.d_pixels,
      left.n_rows, left.n_cols, left.n_slices,
      r_x, r_y, r_max, -offset_x, 0);
  checkCudaErrors(hipGetLastError());
  barrel_distort_ovr<<<gridSize, blockSize>>>(C.d_pixels, right.d_pixels,
      right.n_rows, right.n_cols, right.n_slices,
      r_x, r_y, r_max, offset_x, 1);
  checkCudaErrors(hipGetLastError());

  // resize the image just in case and return
  return gpu_imresize2(C, 800, 1280); // note: resizing is SUPER SLOW
}
