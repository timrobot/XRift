#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <iostream>
#include <cstdlib>
#define checkCudaErrors(val) __check( (val), #val, __FILE__, __LINE__)

template<typename T>
void __check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}

int main() {
	int *data;
	checkCudaErrors(hipMalloc((void **)&data, 4));
	return 0;
}
